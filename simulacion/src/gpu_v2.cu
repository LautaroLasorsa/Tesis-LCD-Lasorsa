#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include "myUtils.cpp"
#include "config.hpp"

namespace GPU_V2{
    
    
    const int SIZE = (INDIVIDUOS_POR_SIMULACION)/32;
    #define LEFT_CHILD(x) ((x) * 2 + 1)
    #define RIGHT_CHILD(x) ((x) * 2 + 2)
    __device__ void heapify(double *array, int n, int i) {
        int largest = i;
        int left = LEFT_CHILD(i);
        int right = RIGHT_CHILD(i);

        if (left < n && array[left] > array[largest]) {
            largest = left;
        }

        if (right < n && array[right] > array[largest]) {
            largest = right;
        }

        if (largest != i) {
            double temp = array[i];
            array[i] = array[largest];
            array[largest] = temp;
            heapify(array, n, largest);
        }
    }

    __device__ void HeapSort(double *f, size_t s) {
        // Build heap (rearrange array)
        for (int i = s / 2 - 1; i >= 0; --i) {
            heapify(f, s, i);
        }

        // One by one extract an element from heap
        for (int i = s - 1; i > 0; --i) {
            // Move current root to end
            double temp = f[0];
            f[0] = f[i];
            f[i] = temp;

            // call max heapify on the reduced heap
            heapify(f, i, 0);
        }
    }

    #undef LEFT_CHILD
    #undef RIGHT_CHILD

    // Merge the ranges vec[0:tam] and vec[tam:2*tam] into vec[0:2*tam]
    __device__ void merge(double * vec, double * aux, size_t tam){
        size_t i = 0, j = tam, k = 0;
        while(i < tam && j < 2*tam){
            if(vec[i] < vec[j]){
                aux[k] = vec[i];
                i++;
            }else{
                aux[k] = vec[j];
                j++;
            }
            k++;
        }
        while(i < tam){
            aux[k] = vec[i];
            i++;
            k++;
        }
        while(j < 2*tam){
            aux[k] = vec[j];
            j++;
            k++;
        }
        for(size_t i = 0; i < 2*tam; i++){
            vec[i] = aux[i];
        }
    }

    __global__ void simularLognormales(double * resultados, int * divisores, double ** muestras, double ** auxiliares) {
        int tid = blockIdx.x;
        int index = tid * (2 * SIMULACIONES_POR_BLOQUE);


    //  double * muestra = new double[INDIVIDUOS_POR_SIMULACION];
    //  double * auxiliar = new double[INDIVIDUOS_POR_SIMULACION];

        double * muestra = muestras[blockIdx.x];
        double * auxiliar = auxiliares[blockIdx.x];

        // printf("Simulacion %d => %p %p\n", tid, muestra, auxiliar);

        for (int s = 0; s < (2 * SIMULACIONES_POR_BLOQUE); ++s) {
            __syncthreads();
            double * resultado = resultados + (index+s) * NUM_DIVISORES;
            hiprandState state;
            hiprand_init(index + s, 0, 0, &state);

            if(threadIdx.x == 0){
                for (int i = 0; i < INDIVIDUOS_POR_SIMULACION; ++i) {
                    muestra[i+1] = hiprand_log_normal(&state, 0.0f, 1.0f);
                //    printf("%f ", muestra[i+1]);
                }
                //printf("\n");
                
                //HeapSort(muestra, INDIVIDUOS_POR_SIMULACION + 1);
                
                //for (int i = 1; i<=INDIVIDUOS_POR_SIMULACION; i++){
                //    muestra[i] = muestra[i] + muestra[i-1];
                // }
            }
            __syncthreads();
            
            if(threadIdx.x<32){
                HeapSort(muestra + 1 + SIZE*threadIdx.x, SIZE);
            }
            
            __syncthreads();

            for(int po2 = 2; po2<=32; po2*=2){
                if(threadIdx.x<32 && threadIdx.x%(po2) == 0){
                    merge(muestra + 1 + SIZE*(threadIdx.x), auxiliar + 1 + SIZE*(threadIdx.x), SIZE*(po2/2));
                }
                __syncthreads();
            }

            if(threadIdx.x == 0){
                for(int i = 0;i<INDIVIDUOS_POR_SIMULACION;i++){
                    assert(muestra[i]<muestra[i+1]+1e-6);
                //    assert(muestra[i]>muestra[i+1]-1e-6);
                }
                for (int i = 1; i<=INDIVIDUOS_POR_SIMULACION; i++){
                    muestra[i] = muestra[i] + muestra[i-1];
                }
            }

            __syncthreads();
            // printf("threadIdx.x = %d\n", threadIdx.x);
            double suma = 0;
            for(int i = 0; i < INDIVIDUOS_POR_SIMULACION; i+=divisores[threadIdx.x]){
                suma += log((muestra[i+divisores[threadIdx.x]] - muestra[i])/divisores[threadIdx.x]);
            }
            resultado[threadIdx.x] = suma/(INDIVIDUOS_POR_SIMULACION/divisores[threadIdx.x]);
            __syncthreads();
        }
    }

    __host__ void SimulacionCuda(){
        int numSimulaciones = (BLOCK_COUNT / 2) * (2 * SIMULACIONES_POR_BLOQUE);
        size_t resultSize = numSimulaciones * NUM_DIVISORES * sizeof(double);

        // Allocate memory on host
        double *h_resultados = (double *)malloc(resultSize);

        // Allocate memory on device
        double *d_resultados;
        hipMalloc(&d_resultados, resultSize);

        // Allocate memory on device for each thread
        double ** h_muestras, ** d_muestras, ** h_auxiliar, ** d_auxiliar;
        h_muestras = (double **)malloc((BLOCK_COUNT / 2) * sizeof(double *));
        h_auxiliar = (double **)malloc((BLOCK_COUNT / 2) * sizeof(double *));

        for (int i = 0; i < (BLOCK_COUNT / 2); ++i) {
            hipMalloc((void **)&h_muestras[i], (INDIVIDUOS_POR_SIMULACION + 1) * sizeof(double));
            hipMalloc((void **)&h_auxiliar[i], (INDIVIDUOS_POR_SIMULACION + 1) * sizeof(double));
        }

        hipMalloc((void **)&d_muestras, (BLOCK_COUNT / 2) * sizeof(double *));
        hipMalloc((void **)&d_auxiliar, (BLOCK_COUNT / 2) * sizeof(double *));
        
        hipMemcpy(d_muestras, h_muestras, (BLOCK_COUNT / 2) * sizeof(double *), hipMemcpyHostToDevice);
        hipMemcpy(d_auxiliar, h_auxiliar, (BLOCK_COUNT / 2) * sizeof(double *), hipMemcpyHostToDevice);
        
        // Copy divisors

        int * d_divisores;
        hipMalloc(&d_divisores, NUM_DIVISORES * sizeof(int));
        hipMemcpy(d_divisores, DIVISORES, NUM_DIVISORES * sizeof(int), hipMemcpyHostToDevice);

        // Launch kernel
        simularLognormales<<<(BLOCK_COUNT / 2), NUM_DIVISORES>>>(d_resultados, d_divisores, d_muestras, d_auxiliar);

        // Copy results from device to host
        hipMemcpy(h_resultados, d_resultados, resultSize, hipMemcpyDeviceToHost);

        exportCSV(h_resultados, DIVISORES, NUM_DIVISORES, NUM_DIVISORES * numSimulaciones, "datos_gpu_v2.csv");

        // Free memory
        free(h_resultados);
        hipFree(d_resultados);
        for (int i = 0; i < (BLOCK_COUNT / 2); ++i) {
            hipFree(h_muestras[i]);
            hipFree(h_auxiliar[i]);
        }
        free(h_muestras);
        free(h_auxiliar);
        hipFree(d_muestras);
        hipFree(d_auxiliar);
        hipFree(d_divisores);
        
    }
}