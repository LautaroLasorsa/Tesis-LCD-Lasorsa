#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "myUtils.cpp"
#include "config.hpp"

namespace GPU_V1{
    #define LEFT_CHILD(x) ((x) * 2 + 1)
    #define RIGHT_CHILD(x) ((x) * 2 + 2)
    __device__ void heapify(double *array, int n, int i) {
        int largest = i;
        int left = LEFT_CHILD(i);
        int right = RIGHT_CHILD(i);

        if (left < n && array[left] > array[largest]) {
            largest = left;
            
        }

        if (right < n && array[right] > array[largest]) {
            largest = right;
        }

        if (largest != i) {
            double temp = array[i];
            array[i] = array[largest];
            array[largest] = temp;
            heapify(array, n, largest);
        }
    }

    __device__ void HeapSort(double *f, size_t s) {
        // Build heap (rearrange array)
        for (int i = s / 2 - 1; i >= 0; --i) {
            heapify(f, s, i);
        }

        // One by one extract an element from heap
        for (int i = s - 1; i > 0; --i) {
            // Move current root to end
            double temp = f[0];
            f[0] = f[i];
            f[i] = temp;

            // call max heapify on the reduced heap
            heapify(f, i, 0);
        }
    }

    #undef LEFT_CHILD
    #undef RIGHT_CHILD

    __global__ void simularLognormales(double * resultados, int * divisores, double ** muestras) {
        int tid = blockIdx.x;
        int index = tid * SIMULACIONES_POR_BLOQUE;


    //  double * muestra = new double[INDIVIDUOS_POR_SIMULACION];
    //  double * auxiliar = new double[INDIVIDUOS_POR_SIMULACION];

        double * muestra = muestras[blockIdx.x];
        
        // printf("Simulacion %d => %p %p\n", tid, muestra, auxiliar);

        for (int s = 0; s < SIMULACIONES_POR_BLOQUE; ++s) {
            __syncthreads();
            double * resultado = resultados + (index+s) * NUM_DIVISORES;
            hiprandState state;
            hiprand_init(index + s, 0, 0, &state);

            if(threadIdx.x == 0){
                for (int i = 0; i < INDIVIDUOS_POR_SIMULACION; ++i) {
                    muestra[i+1] = hiprand_log_normal(&state, 0.0f, 1.0f);
                //    printf("%f ", muestra[i+1]);
                }
                //printf("\n");
                
                HeapSort(muestra, INDIVIDUOS_POR_SIMULACION + 1);
                
                for (int i = 1; i<=INDIVIDUOS_POR_SIMULACION; i++){
                    muestra[i] = muestra[i] + muestra[i-1];
                }
            }
            __syncthreads();
            // printf("threadIdx.x = %d\n", threadIdx.x);
            double suma = 0;
            for(int i = 0; i < INDIVIDUOS_POR_SIMULACION; i+=divisores[threadIdx.x]){
                suma += log((muestra[i+divisores[threadIdx.x]] - muestra[i])/divisores[threadIdx.x]);
            }
            resultado[threadIdx.x] = suma/(INDIVIDUOS_POR_SIMULACION/divisores[threadIdx.x]);
            __syncthreads();
        }
    }

    __host__ void SimulacionCuda(){
        const int numSimulaciones = BLOCK_COUNT * SIMULACIONES_POR_BLOQUE;
        size_t resultSize = numSimulaciones * NUM_DIVISORES * sizeof(double);

        // Allocate memory on host
        double *h_resultados = (double *)malloc(resultSize);

        // Allocate memory on device
        double *d_resultados;
        hipMalloc(&d_resultados, resultSize);

        // Allocate memory on device for each thread
        double ** h_muestras, ** d_muestras;
        h_muestras = (double **)malloc(BLOCK_COUNT * sizeof(double *));

        for (int i = 0; i < BLOCK_COUNT; ++i) {
            hipMalloc((void **)&h_muestras[i], (INDIVIDUOS_POR_SIMULACION + 1) * sizeof(double));
        }

        hipMalloc((void **)&d_muestras, BLOCK_COUNT * sizeof(double *));
        hipMemcpy(d_muestras, h_muestras, BLOCK_COUNT * sizeof(double *), hipMemcpyHostToDevice);

        // Copy divisors

        int * d_divisores;
        hipMalloc(&d_divisores, NUM_DIVISORES * sizeof(int));
        hipMemcpy(d_divisores, DIVISORES, NUM_DIVISORES * sizeof(int), hipMemcpyHostToDevice);

        // Launch kernel
        simularLognormales<<<BLOCK_COUNT, NUM_DIVISORES>>>(d_resultados, d_divisores, d_muestras);

        // Copy results from device to host
        hipMemcpy(h_resultados, d_resultados, resultSize, hipMemcpyDeviceToHost);

        exportCSV(h_resultados, DIVISORES, NUM_DIVISORES, NUM_DIVISORES * numSimulaciones, "datos_gpu_v1.csv");

        // Free memory
        free(h_resultados);
        hipFree(d_resultados);
        for (int i = 0; i < BLOCK_COUNT; ++i) {
            hipFree(h_muestras[i]);
        }
        free(h_muestras);
        hipFree(d_muestras);
        hipFree(d_divisores);
        
    }
}