#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu_v1.cu"
#include "gpu_v2.cu"
#include "gpu_v3.cu"
#include "gpu_varianzas.cu"
#include "cpu_v1.cpp"
#include "cpu_v2.cpp"



// Given a host function measures the execution time and prints it
void MeasureTime(void (*f)(), const std::string& name) {
    clock_t start = clock();
    f();
    clock_t end = clock();
    printf("%s: %f\n", name.c_str(), (double)(end - start) / CLOCKS_PER_SEC);
}


int main(int argc, char **argv) {

//    MeasureTime(CPU_V2::SimulacionCPU, "CPU V2");
//    MeasureTime(GPU_V3::SimulacionCuda, "GPU V3");
//    MeasureTime(GPU_V2::SimulacionCuda, "GPU V2");
//    MeasureTime(GPU_V1::SimulacionCuda, "GPU V1");
//    MeasureTime(CPU_V1::SimulacionCPU, "CPU V1");
//    MeasureTime(GPU_VARIANZAS::SimulacionCuda,"GPU VARIANZAS");
    MeasureTime(GPU_VARIANZAS::SimulacionCuda,std::string(argv[1]));
    return 0;
}
